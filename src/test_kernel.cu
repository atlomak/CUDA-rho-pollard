#include "hip/hip_runtime.h"
#include "bignum.cu"
#include "bn_ec_point_ops.cu"
#include "utils.cuh"

__global__ __launch_bounds__(512, 2) void ker_add_points(EC_parameters *parameters, int32_t instances, EC_point *points)
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= instances)
    {
        return;
    }

    EC_point a, b, c;
    bn Pmod;

    bignum_init(&a.x);
    bignum_init(&a.y);

    bignum_init(&b.x);
    bignum_init(&b.y);

    bignum_init(&c.x);
    bignum_init(&c.y);

    bignum_init(&Pmod);

    bignum_assign(&Pmod, &parameters->Pmod);
    bignum_assign(&a.x, &points[idx * 2].x);
    bignum_assign(&a.y, &points[idx * 2].y);

    bignum_assign(&b.x, &points[idx * 2 + 1].x);
    bignum_assign(&b.y, &points[idx * 2 + 1].y);

    bn temp;
    bn sub;
    bignum_sub(&b.x, &a.x, &sub);
    if (bignum_cmp(&b.x, &a.x) == SMALLER)
    {
        bignum_add(&sub, &Pmod, &temp);
        bignum_assign(&sub, &temp);
    }

    bignum_modinv(&sub, &Pmod, &temp);

    for (int i = 0; i < 10000; i++)
    {
        add_points(&a, &b, &c, &Pmod, &temp);
    }

    bignum_assign(&points[idx * 2].x, &c.x);
    bignum_assign(&points[idx * 2].y, &c.y);
}

extern "C" {
void test_adding_points(EC_point *points, int32_t instances, EC_parameters *parameters)
{
    EC_point *gpuPoints = nullptr;
    EC_parameters *gpuParameters = nullptr;

    hipError_t err;

    hipMalloc(&gpuPoints, instances * 2 * sizeof(EC_point));
    cudaCheckErrors("Failed to allocate device memory for points");

    hipMalloc(&gpuParameters, sizeof(EC_parameters));
    cudaCheckErrors("Failed to allocate device memory for parameters");

    hipMemcpy(gpuPoints, points, instances * 2 * sizeof(EC_point), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy points from host to device");

    hipMemcpy(gpuParameters, parameters, sizeof(EC_parameters), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy parameters from host to device");

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // 16 instances per block, instance = 4 threads
    ker_add_points<<<(instances + 511) / 512, 512>>>(gpuParameters, instances, gpuPoints);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time: %3.1f\n", milliseconds);

    hipGetLastError();
    cudaCheckErrors("Failed to launch kernel");

    hipMemcpy(points, gpuPoints, instances * 2 * sizeof(EC_point), hipMemcpyDeviceToHost);
    cudaCheckErrors("Failed to copy points from device to host");

    hipFree(gpuPoints);
    hipFree(gpuParameters);
}
}
