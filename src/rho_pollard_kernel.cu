#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include "ec_points_ops.cu"

#define PRECOMPUTED_POINTS 1024
#define BATCH_SIZE 6

__shared__ PCMP_point SMEMprecomputed[PRECOMPUTED_POINTS];

__device__ uint32_t is_distinguish(env192_t &bn_env, const dev_EC_point &P, uint32_t zeros_count) { return (cgbn_ctz(bn_env, P.x) >= zeros_count); }

__device__ uint32_t map_to_index(env192_t &bn_env, const dev_EC_point &P, const env192_t::cgbn_t &mask)
{
    env192_t::cgbn_t t;
    cgbn_bitwise_and(bn_env, t, P.x, mask);
    return cgbn_get_ui32(bn_env, t);
}

typedef struct
{
    EC_point *starting;
    PCMP_point *precomputed;
    EC_parameters *parameters;
    uint32_t instances;
    uint32_t n;
} rho_pollard_args;

__global__ __launch_bounds__(256, 2) void rho_pollard(cgbn_error_report_t *report, rho_pollard_args args)
{
    uint32_t instance;
    uint32_t thread_id;

    instance = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
    thread_id = (blockIdx.x * blockDim.x + threadIdx.x);

    if (instance >= args.instances)
    {
        return;
    }


    if (threadIdx.x == 0)
    {
        printf("block: %d\n", blockIdx.x);
        for (int i = 0; i < PRECOMPUTED_POINTS; i++)
        {
            SMEMprecomputed[i] = args.precomputed[i];
        }
    }

    __syncthreads();

    context_t bn_context(cgbn_report_monitor, report, instance); // construct a context
    env192_t bn192_env(bn_context.env<env192_t>());

    dev_Parameters params;


    env192_t::cgbn_t mask;


    cgbn_load(bn192_env, params.Pmod, &(args.parameters->Pmod));
    cgbn_load(bn192_env, params.a, &(args.parameters->a));
    params.clz_count = cgbn_barrett_approximation(bn192_env, params.approx, params.Pmod);

    cgbn_set_ui32(bn192_env, mask, PRECOMPUTED_POINTS - 1);


    env192_t::cgbn_t b[BATCH_SIZE];
    dev_EC_point W[BATCH_SIZE], R[BATCH_SIZE];

    int read_offset;
    for (int i = 0; i < BATCH_SIZE; i++)
    {
        read_offset = i * args.instances;
        cgbn_load(bn192_env, W[i].x, &(args.starting[instance + read_offset].x));
        cgbn_load(bn192_env, W[i].y, &(args.starting[instance + read_offset].y));
        cgbn_load(bn192_env, W[i].seed, &(args.starting[instance + read_offset].seed));
    }
    read_offset += args.instances; // Dont read from same offset twice

    int counter = 0;
    int found_flags[BATCH_SIZE] = {0};
    while (counter < args.n)
    {

        for (int i = 0; i < BATCH_SIZE; i++)
        {
            uint32_t precom_index = map_to_index(bn192_env, W[i], mask);
            cgbn_load(bn192_env, R[i].x, &(SMEMprecomputed[precom_index].x));
            cgbn_load(bn192_env, R[i].y, &(SMEMprecomputed[precom_index].y));
        }

        env192_t::cgbn_t a[BATCH_SIZE];
        for (int i = 0; i < BATCH_SIZE; i++)
        {
            if (cgbn_sub(bn192_env, a[i], W[i].x, R[i].x))
            {
                cgbn_add(bn192_env, a[i], a[i], params.Pmod);
            }
        }

        env192_t::cgbn_t v;
        cgbn_set_ui32(bn192_env, v, 1);

        for (int i = 0; i < BATCH_SIZE; i++)
        {
            env192_t::cgbn_wide_t wide;
            cgbn_set(bn192_env, b[i], v);
            cgbn_mul_wide(bn192_env, wide, v, a[i]);
            cgbn_barrett_rem_wide(bn192_env, v, wide, params.Pmod, params.approx, params.clz_count);
        }

        env192_t::cgbn_t x;
        cgbn_modular_inverse(bn192_env, x, v, params.Pmod);

        for (int i = BATCH_SIZE - 1; i >= 0; i--)
        {
            env192_t::cgbn_wide_t wide;
            cgbn_mul_wide(bn192_env, wide, x, b[i]);
            cgbn_barrett_rem_wide(bn192_env, b[i], wide, params.Pmod, params.approx, params.clz_count);

            cgbn_mul_wide(bn192_env, wide, x, a[i]);
            cgbn_barrett_rem_wide(bn192_env, x, wide, params.Pmod, params.approx, params.clz_count);
        }

        for (int i = 0; i < BATCH_SIZE; i++)
        {
            if (found_flags[i] != 1)
            {
                add_points(bn192_env, W[i], W[i], R[i], params, b[i]);
            }
        }

        for (int i = 0; i < BATCH_SIZE; i++)
        {
            if (found_flags[i] != 1 && is_distinguish(bn192_env, W[i], args.parameters->zeros_count))
            {
                int offset;
                offset = counter * args.instances;
                cgbn_store(bn192_env, &(args.starting[instance + offset].x), W[i].x);
                cgbn_store(bn192_env, &(args.starting[instance + offset].y), W[i].y);
                cgbn_store(bn192_env, &(args.starting[instance + offset].seed), W[i].seed);
                counter++;
                if (thread_id % TPI == 0)
                {
                    printf("counter %d ,Instance %d found distinguish point!\n", counter, instance);
                }
                found_flags[i] = 1;
                if (read_offset < args.instances * args.n)
                {
                    cgbn_load(bn192_env, W[i].x, &(args.starting[instance + read_offset].x));
                    cgbn_load(bn192_env, W[i].y, &(args.starting[instance + read_offset].y));
                    cgbn_load(bn192_env, W[i].seed, &(args.starting[instance + read_offset].seed));
                    read_offset += args.instances;
                    found_flags[i] = 0;
                }
            }
        }
    }
}

extern "C" {
void run_rho_pollard(EC_point *startingPts, uint32_t instances, uint32_t n, PCMP_point *precomputed_points, EC_parameters *parameters)
{
    printf("Starting rho pollard: zeroes count %d", parameters->zeros_count);
    EC_point *gpu_starting;
    PCMP_point *gpu_precomputed;
    EC_parameters *gpu_params;
    cgbn_error_report_t *report;

    hipSetDevice(0);
    cudaCheckErrors("Failed to set device");

    hipHostMalloc((void **)&gpu_starting, sizeof(EC_point) * instances * n);
    cudaCheckErrors("Failed to allocate memory for starting points");

    hipMemcpy(gpu_starting, startingPts, sizeof(EC_point) * instances * n, hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy starting points to device");

    hipMalloc((void **)&gpu_precomputed, sizeof(PCMP_point) * PRECOMPUTED_POINTS);
    cudaCheckErrors("Failed to allocate memory for precomputed points");

    hipMemcpy(gpu_precomputed, precomputed_points, sizeof(PCMP_point) * PRECOMPUTED_POINTS, hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy precomputed points to device");

    hipMalloc((void **)&gpu_params, sizeof(EC_parameters));
    cudaCheckErrors("Failed to allocate memory for parameters");

    hipMemcpy(gpu_params, parameters, sizeof(EC_parameters), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy parameters to device");

    rho_pollard_args args;
    args.starting = gpu_starting;
    args.precomputed = gpu_precomputed;
    args.parameters = gpu_params;
    args.instances = instances;
    args.n = n;

    cgbn_error_report_alloc(&report);
    cudaCheckErrors("Failed to allocate memory for error report");

    int numBlocks; // Occupancy in terms of active blocks
    int blockSize = 256;

    // These variables are used to convert occupancy to warps
    int device;
    hipDeviceProp_t prop;
    int activeWarps;
    int maxWarps;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, rho_pollard, blockSize, 0);

    activeWarps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    printf("Active warps: %d\n", activeWarps);
    printf("Occupancy: %f\n", (double)activeWarps / maxWarps);

    hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, rho_pollard, 0, 0);

    printf("Max potential block size: %d\n", blockSize);

    // 512 threads per block (128 CGBN instances)
    rho_pollard<<<(instances + 7) / 8, 256>>>(report, args);

    hipDeviceSynchronize();
    cudaCheckErrors("Kernel failed");

    CGBN_CHECK(report);


    hipMemcpy(startingPts, gpu_starting, sizeof(EC_point) * instances * n, hipMemcpyDeviceToHost);
    cudaCheckErrors("Failed to copy starting points to host");

    hipHostFree(gpu_starting);
    cudaCheckErrors("Failed to free memory for starting points");

    hipFree(gpu_precomputed);
    cudaCheckErrors("Failed to free memory for precomputed points");

    hipFree(gpu_params);
    cudaCheckErrors("Failed to free memory for parameters");

    cgbn_error_report_free(report);
    cudaCheckErrors("Failed to free memory for error report");
}
}
