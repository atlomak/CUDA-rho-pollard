#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include "ec_points_ops.cu"

#define PRECOMPUTED_POINTS 1024

__shared__ EC_point SMEMprecomputed[PRECOMPUTED_POINTS];

__device__ uint32_t is_distinguish(env192_t &bn_env, const dev_EC_point &P, uint32_t zeros_count) { return (cgbn_ctz(bn_env, P.x) >= zeros_count); }

__device__ uint32_t map_to_index(env192_t &bn_env, const dev_EC_point &P, const env192_t::cgbn_t &mask)
{
    env192_t::cgbn_t t;
    cgbn_bitwise_and(bn_env, t, P.x, mask);
    return cgbn_get_ui32(bn_env, t);
}

typedef struct
{
    EC_point *starting;
    EC_point *precomputed;
    EC_parameters *parameters;
    uint32_t instances;
    uint32_t n;
} rho_pollard_args;

__global__ void rho_pollard(cgbn_error_report_t *report, rho_pollard_args args)
{
    uint32_t instance;
    uint32_t thread_id;

    instance = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
    thread_id = (blockIdx.x * blockDim.x + threadIdx.x);

    if (instance >= args.instances)
    {
        return;
    }


    if (threadIdx.x == 0)
    {
        printf("block: %d\n", blockIdx.x);
        for (int i = 0; i < PRECOMPUTED_POINTS; i++)
        {
            SMEMprecomputed[i] = args.precomputed[i];
        }
    }

    __syncthreads();

    context_t bn_context(cgbn_report_monitor, report, instance); // construct a context
    env192_t bn192_env(bn_context.env<env192_t>());

    dev_EC_point W, R;
    dev_Parameters params;


    env192_t::cgbn_t mask;


    cgbn_load(bn192_env, params.Pmod, &(args.parameters->Pmod));
    cgbn_load(bn192_env, params.a, &(args.parameters->a));
    params.clz_count = cgbn_barrett_approximation(bn192_env, params.approx, params.Pmod);

    cgbn_set_ui32(bn192_env, mask, PRECOMPUTED_POINTS - 1);

    int offset;
    for (int i = 0; i < args.n; i++)
    {
        offset = i * args.instances;
        cgbn_load(bn192_env, W.x, &(args.starting[instance + offset].x));
        cgbn_load(bn192_env, W.y, &(args.starting[instance + offset].y));

        uint32_t counter = 0;
        while (!is_distinguish(bn192_env, W, args.parameters->zeros_count))
        {
            counter++;
            uint32_t precomp_index = map_to_index(bn192_env, W, mask);
            cgbn_load(bn192_env, R.x, &(SMEMprecomputed[precomp_index].x));
            cgbn_load(bn192_env, R.y, &(SMEMprecomputed[precomp_index].y));
            add_points(bn192_env, W, W, R, params);
        }
        if (thread_id % TPI == 0)
        {
            printf("%d ,Instance %d found distinguish point, after %d iterations!\n", i, instance, counter);
        }

        cgbn_store(bn192_env, &(args.starting[instance + offset].x), W.x);
        cgbn_store(bn192_env, &(args.starting[instance + offset].y), W.y);
    }
}

extern "C" {
void run_rho_pollard(EC_point *startingPts, uint32_t instances, uint32_t n, EC_point *precomputed_points, EC_parameters *parameters)
{
    printf("Starting rho pollard: zeroes count %d", parameters->zeros_count);
    EC_point *gpu_starting;
    EC_point *gpu_precomputed;
    EC_parameters *gpu_params;
    cgbn_error_report_t *report;

    hipSetDevice(0);
    cudaCheckErrors("Failed to set device");

    hipHostMalloc((void **)&gpu_starting, sizeof(EC_point) * instances * n);
    cudaCheckErrors("Failed to allocate memory for starting points");

    hipMemcpy(gpu_starting, startingPts, sizeof(EC_point) * instances * n, hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy starting points to device");

    hipMalloc((void **)&gpu_precomputed, sizeof(EC_point) * PRECOMPUTED_POINTS);
    cudaCheckErrors("Failed to allocate memory for precomputed points");

    hipMemcpy(gpu_precomputed, precomputed_points, sizeof(EC_point) * PRECOMPUTED_POINTS, hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy precomputed points to device");

    hipMalloc((void **)&gpu_params, sizeof(EC_parameters));
    cudaCheckErrors("Failed to allocate memory for parameters");

    hipMemcpy(gpu_params, parameters, sizeof(EC_parameters), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy parameters to device");

    rho_pollard_args args;
    args.starting = gpu_starting;
    args.precomputed = gpu_precomputed;
    args.parameters = gpu_params;
    args.instances = instances;
    args.n = n;

    cgbn_error_report_alloc(&report);
    cudaCheckErrors("Failed to allocate memory for error report");

    // 512 threads per block (128 CGBN instances)
    rho_pollard<<<(instances + 63) / 64, 256>>>(report, args);

    hipDeviceSynchronize();
    cudaCheckErrors("Kernel failed");

    CGBN_CHECK(report);


    hipMemcpy(startingPts, gpu_starting, sizeof(EC_point) * instances * n, hipMemcpyDeviceToHost);
    cudaCheckErrors("Failed to copy starting points to host");

    hipFree(gpu_starting);
    cudaCheckErrors("Failed to free memory for starting points");

    hipFree(gpu_precomputed);
    cudaCheckErrors("Failed to free memory for precomputed points");

    hipFree(gpu_params);
    cudaCheckErrors("Failed to free memory for parameters");

    cgbn_error_report_free(report);
    cudaCheckErrors("Failed to free memory for error report");
}
}
