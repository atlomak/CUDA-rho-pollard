#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include "ec_points_ops.cu"

#define PRECOMPUTED_POINTS 850
#define BATCH_SIZE 5

__shared__ PCMP_point SMEMprecomputed[PRECOMPUTED_POINTS];

__shared__ int warp_finished;

__device__ uint32_t is_distinguish(env192_t &bn_env, env192_t::cgbn_t &x, uint32_t zeros_count) { return (cgbn_ctz(bn_env, x) >= zeros_count); }

__device__ uint32_t map_to_index(env192_t &bn_env, env192_t::cgbn_t &x, const env192_t::cgbn_t &mask)
{
    env192_t::cgbn_t t;
    cgbn_bitwise_and(bn_env, t, x, mask);
    return cgbn_get_ui32(bn_env, t);
}

typedef struct
{
    EC_point *starting;
    PCMP_point *precomputed;
    EC_parameters *parameters;
    uint32_t instances;
    uint32_t n;
    int stream;
} rho_pollard_args;

__global__ __launch_bounds__(224, 3) void rho_pollard(cgbn_error_report_t *report, rho_pollard_args args, int stream)
{
    uint32_t instance;
    uint32_t thread_id;

    instance = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
    thread_id = (blockIdx.x * blockDim.x + threadIdx.x);

    if (instance >= args.instances)
    {
        return;
    }


    if (threadIdx.x == 0)
    {
        printf("block: %d\n", blockIdx.x);
        for (int i = 0; i < PRECOMPUTED_POINTS; i++)
        {
            SMEMprecomputed[i] = args.precomputed[i];
        }
        warp_finished = 0;
    }

    __syncthreads();

    context_t bn_context(cgbn_report_monitor, report, instance); // construct a context
    env192_t bn192_env(bn_context.env<env192_t>());

    dev_Parameters params;


    env192_t::cgbn_t mask;


    cgbn_load(bn192_env, params.Pmod, &(args.parameters->Pmod));
    cgbn_load(bn192_env, params.a, &(args.parameters->a));
    params.clz_count = cgbn_barrett_approximation(bn192_env, params.approx, params.Pmod);

    cgbn_set_ui32(bn192_env, mask, PRECOMPUTED_POINTS - 1);


    env192_t::cgbn_t b[BATCH_SIZE];
    dev_EC_point_local W[BATCH_SIZE];

    int read_offset;
    for (int i = 0; i < BATCH_SIZE; i++)
    {
        read_offset = i;
        dev_EC_point P;
        cgbn_load(bn192_env, P.x, &(args.starting[instance * args.n + read_offset].x));
        cgbn_load(bn192_env, P.y, &(args.starting[instance * args.n + read_offset].y));
        cgbn_load(bn192_env, P.seed, &(args.starting[instance * args.n + read_offset].seed));
        cgbn_store(bn192_env, &W[i].x, P.x);
        cgbn_store(bn192_env, &W[i].y, P.y);
        cgbn_store(bn192_env, &W[i].seed, P.seed);
    }
    read_offset += 1; // Dont read from same offset twice

    int counter = 0;
    int found_flags[BATCH_SIZE] = {0};
    while (counter < args.n && warp_finished == 0)
    {

        env192_t::cgbn_t a[BATCH_SIZE];

        for (int i = 0; i < BATCH_SIZE; i++)
        {
            if (found_flags[i] == 1)
            {
                continue;
            }
            env192_t::cgbn_t Px, Rx;
            cgbn_load(bn192_env, Px, &(W[i].x));

            uint32_t precom_index = map_to_index(bn192_env, Px, mask);
            cgbn_load(bn192_env, Rx, &(SMEMprecomputed[precom_index].x));

            if (cgbn_sub(bn192_env, a[i], Px, Rx))
            {
                cgbn_add(bn192_env, a[i], a[i], params.Pmod);
            }
        }

        env192_t::cgbn_t v;
        cgbn_set_ui32(bn192_env, v, 1);

        for (int i = 0; i < BATCH_SIZE; i++)
        {
            if (found_flags[i] == 1)
            {
                continue;
            }
            env192_t::cgbn_wide_t wide;
            cgbn_set(bn192_env, b[i], v);
            cgbn_mul_wide(bn192_env, wide, v, a[i]);
            cgbn_barrett_rem_wide(bn192_env, v, wide, params.Pmod, params.approx, params.clz_count);
        }

        env192_t::cgbn_t x;
        cgbn_modular_inverse(bn192_env, x, v, params.Pmod);

        for (int i = BATCH_SIZE - 1; i >= 0; i--)
        {
            if (found_flags[i] == 1)
            {
                continue;
            }
            env192_t::cgbn_wide_t wide;
            cgbn_mul_wide(bn192_env, wide, x, b[i]);
            cgbn_barrett_rem_wide(bn192_env, b[i], wide, params.Pmod, params.approx, params.clz_count);

            cgbn_mul_wide(bn192_env, wide, x, a[i]);
            cgbn_barrett_rem_wide(bn192_env, x, wide, params.Pmod, params.approx, params.clz_count);
        }

        for (int i = 0; i < BATCH_SIZE; i++)
        {
            dev_EC_point R, P;
            if (found_flags[i] != 1)
            {
                cgbn_load(bn192_env, P.x, &(W[i].x));
                cgbn_load(bn192_env, P.y, &(W[i].y));
                cgbn_load(bn192_env, P.seed, &(W[i].seed));

                uint32_t precom_index = map_to_index(bn192_env, P.x, mask);
                cgbn_load(bn192_env, R.x, &(SMEMprecomputed[precom_index].x));
                cgbn_load(bn192_env, R.y, &(SMEMprecomputed[precom_index].y));
                add_points(bn192_env, P, P, R, params, b[i]);

                cgbn_store(bn192_env, &W[i].x, P.x);
                cgbn_store(bn192_env, &W[i].y, P.y);
                cgbn_store(bn192_env, &W[i].seed, P.seed);
            }

            if (found_flags[i] != 1 && is_distinguish(bn192_env, P.x, args.parameters->zeros_count))
            {
                cgbn_store(bn192_env, &(args.starting[instance * args.n + counter].x), P.x);
                cgbn_store(bn192_env, &(args.starting[instance * args.n + counter].y), P.y);
                cgbn_store(bn192_env, &(args.starting[instance * args.n + counter].seed), P.seed);
                counter++;
                if (thread_id % TPI == 0)
                {
                    printf("STREAM: %d, counter %d ,Instance %d found distinguish point!\n", stream, counter, instance);
                }
                found_flags[i] = 1;
                if (read_offset < args.n)
                {
                    cgbn_load(bn192_env, P.x, &(args.starting[instance * args.n + read_offset].x));
                    cgbn_load(bn192_env, P.y, &(args.starting[instance * args.n + read_offset].y));
                    cgbn_load(bn192_env, P.seed, &(args.starting[instance * args.n + read_offset].seed));

                    cgbn_store(bn192_env, &W[i].x, P.x);
                    cgbn_store(bn192_env, &W[i].y, P.y);
                    cgbn_store(bn192_env, &W[i].seed, P.seed);

                    read_offset += 1;
                    found_flags[i] = 0;
                }
            }
        }
    }
    warp_finished = 1;
}

extern "C" {
void run_rho_pollard(EC_point *startingPts, uint32_t instances, uint32_t n, PCMP_point *precomputed_points, EC_parameters *parameters)
{
    printf("Starting rho pollard: zeroes count %d", parameters->zeros_count);
    EC_point *gpu_starting;
    PCMP_point *gpu_precomputed;
    EC_parameters *gpu_params;
    cgbn_error_report_t *report;

    hipSetDevice(0);
    cudaCheckErrors("Failed to set device");

    hipHostMalloc((void **)&gpu_starting, sizeof(EC_point) * instances * n);
    cudaCheckErrors("Failed to allocate memory for starting points");

    hipMemcpy(gpu_starting, startingPts, sizeof(EC_point) * instances * n, hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy starting points to device");

    hipMalloc((void **)&gpu_precomputed, sizeof(PCMP_point) * PRECOMPUTED_POINTS);
    cudaCheckErrors("Failed to allocate memory for precomputed points");

    hipMemcpy(gpu_precomputed, precomputed_points, sizeof(PCMP_point) * PRECOMPUTED_POINTS, hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy precomputed points to device");

    hipMalloc((void **)&gpu_params, sizeof(EC_parameters));
    cudaCheckErrors("Failed to allocate memory for parameters");

    hipMemcpy(gpu_params, parameters, sizeof(EC_parameters), hipMemcpyHostToDevice);
    cudaCheckErrors("Failed to copy parameters to device");

    rho_pollard_args args;
    args.starting = gpu_starting;
    args.precomputed = gpu_precomputed;
    args.parameters = gpu_params;
    args.instances = instances;
    args.n = n;

    cgbn_error_report_alloc(&report);
    cudaCheckErrors("Failed to allocate memory for error report");

    int numBlocks; // Occupancy in terms of active blocks
    int blockSize = 512;

    // These variables are used to convert occupancy to warps
    int device;
    hipDeviceProp_t prop;
    int activeWarps;
    int maxWarps;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, rho_pollard, blockSize, 0);

    activeWarps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    printf("Active warps: %d\n", activeWarps);
    printf("Occupancy: %f\n", (double)activeWarps / maxWarps);

    hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, rho_pollard, 0, 0);

    printf("Max potential block size: %d\n", blockSize);

    // 512 threads per block (128 CGBN instances)
    rho_pollard<<<(instances + 11) / 12, 384>>>(report, args);

    hipDeviceSynchronize();
    cudaCheckErrors("Kernel failed");

    CGBN_CHECK(report);


    hipMemcpy(startingPts, gpu_starting, sizeof(EC_point) * instances * n, hipMemcpyDeviceToHost);
    cudaCheckErrors("Failed to copy starting points to host");

    hipHostFree(gpu_starting);
    cudaCheckErrors("Failed to free memory for starting points");

    hipFree(gpu_precomputed);
    cudaCheckErrors("Failed to free memory for precomputed points");

    hipFree(gpu_params);
    cudaCheckErrors("Failed to free memory for parameters");

    cgbn_error_report_free(report);
    cudaCheckErrors("Failed to free memory for error report");
}
}
