#include "hip/hip_runtime.h"
//
// Created by atlomak on 05.05.24.
//

#include "main.cuh"

__device__ dev_ECC_192_point add_points(env192_t bn_env, const dev_ECC_192_point &P1, const dev_ECC_192_point &P2, const env192_t::cgbn_t &Pmod)
{
    env192_t::cgbn_t t2;
    if (cgbn_sub(bn_env, t2, P1.x, P2.x)) // x1 - x2 mod Pmod
    {
        cgbn_sub(bn_env, t2, P2.x, P1.x);
        cgbn_sub(bn_env, t2, Pmod, t2);
    }

    cgbn_modular_inverse(bn_env, t2, t2, Pmod); // 1/(x1-x2) mod Pmod

    // Montgomery space

    env192_t::cgbn_t x1, y1, x2, y2;

    uint32_t np0;
    np0 = cgbn_bn2mont(bn_env, x1, P1.x, Pmod);
    cgbn_bn2mont(bn_env, y1, P1.y, Pmod);
    cgbn_bn2mont(bn_env, x2, P2.x, Pmod);
    cgbn_bn2mont(bn_env, y2, P2.y, Pmod);
    cgbn_bn2mont(bn_env, t2, t2, Pmod);

    env192_t::cgbn_t t1;

    if (cgbn_sub(bn_env, t1, y1, y2)) // y0 - y1 mod Pmod
    {
        cgbn_sub(bn_env, t1, y2, y1);
        cgbn_sub(bn_env, t1, Pmod, t1);
    }

    env192_t::cgbn_t s, s_sq, x3, y3, t3;

    cgbn_mont_mul(bn_env, s, t1, t2, Pmod, np0); // s = (y1-y2)/(x1-x2) mod Pmod // tested
    cgbn_mont_sqr(bn_env, s_sq, s, Pmod, np0); // s^2 mod Pmod // tested

    cgbn_add(bn_env, t3, x1, x2); // x1 + x2

    if (cgbn_sub(bn_env, x3, s_sq, t3)) // x3 = s^2 - x1 - x2 // mod Pmod
    {
        cgbn_sub(bn_env, x3, t3, s_sq);
        cgbn_sub(bn_env, x3, Pmod, x3);
    }

    if (cgbn_sub(bn_env, t3, x1, x3)) // t3 = x1 - x3 // mod Pmod
    {
        cgbn_sub(bn_env, t3, x3, x1);
        cgbn_sub(bn_env, t3, Pmod, t3);
    }

    cgbn_mont_mul(bn_env, t3, t3, s, Pmod, np0);

    if (cgbn_sub(bn_env, y3, t3, y1))
    {
        cgbn_sub(bn_env, y3, y1, t3);
        cgbn_sub(bn_env, y3, Pmod, y3);
    }

    cgbn_mont2bn(bn_env, x3, x3, Pmod, np0);
    cgbn_mont2bn(bn_env, y3, y3, Pmod, np0);

    // cgbn_sub(bn_env, x3, s_sq, temp);

    return dev_ECC_192_point{x3, y3};
}

__device__ dev_ECC_192_point point_mul(env192_t bn_env, const dev_ECC_192_point &P, const env192_t::cgbn_t &k, const env192_t::cgbn_t &Pmod)
{
    dev_ECC_192_point R = P;
    dev_ECC_192_point Q = P;

    for (int i = 1; i < 192; i++)
    {
        R = add_points(bn_env, R, Q, Pmod);
    }

    return R;
}

#ifndef UNIT_TESTING
int main() {}
#endif
