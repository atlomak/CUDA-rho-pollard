#include "hip/hip_runtime.h"
//
// Created by atlomak on 05.05.24.
//


#include "catch2/catch_test_macros.hpp"
#include "../src/main.cu"


// Minimal kernel to test device add_points function
__global__ void kernel_test(cgbn_error_report_t *report, ECC_192_point *points, EC_parameters *parameters)
{
    context_t bn_context(cgbn_report_monitor, report, 1); // construct a context
    env192_t bn192_env(bn_context.env<env192_t>());

    env192_t::cgbn_t Pmod;
    dev_ECC_192_point P0, P1;

    cgbn_load(bn192_env, P0.x, &(points[0].x));
    cgbn_load(bn192_env, P0.y, &(points[0].y));

    cgbn_load(bn192_env, P1.x, &(points[1].x));
    cgbn_load(bn192_env, P1.y, &(points[1].y));

    cgbn_load(bn192_env, Pmod, &(parameters->Pmod));

    dev_ECC_192_point result = add_points(bn192_env, P0, P1, Pmod);

    cgbn_store(bn192_env, &(points[0].x), result.x);
    cgbn_store(bn192_env, &(points[0].y), result.y);
}

// minimal kernel to test double point
__global__ void kernel_test_double(cgbn_error_report_t *report, ECC_192_point *points, EC_parameters *parameters)
{
    context_t bn_context(cgbn_report_monitor, report, 1); // construct a context
    env192_t bn192_env(bn_context.env<env192_t>());

    env192_t::cgbn_t Pmod, a;
    dev_ECC_192_point P0, P1;

    cgbn_load(bn192_env, P0.x, &(points[0].x));
    cgbn_load(bn192_env, P0.y, &(points[0].y));

    cgbn_load(bn192_env, P1.x, &(points[1].x));
    cgbn_load(bn192_env, P1.y, &(points[1].y));

    cgbn_load(bn192_env, Pmod, &(parameters->Pmod));
    cgbn_load(bn192_env, a, &(parameters->a));

    dev_ECC_192_point result = double_point(bn192_env, P0, P0, Pmod, a);

    cgbn_store(bn192_env, &(points[0].x), result.x);
    cgbn_store(bn192_env, &(points[0].y), result.y);
}

TEST_CASE("ECC_79p add points [1]")
{
    ECC_192_point *points, *gpuPoints;
    EC_parameters *parameters, *gpuParameters;
    cgbn_error_report_t *report;

    points = (ECC_192_point *)malloc(sizeof(ECC_192_point) * 2);
    parameters = (EC_parameters *)malloc(sizeof(EC_parameters));

    // POINT A
    points[0].x._limbs[0] = 0x8475057d;
    points[0].x._limbs[1] = 0x4b201c20;
    points[0].x._limbs[2] = 0x0000315d;
    points[0].x._limbs[3] = 0x0;
    points[0].x._limbs[4] = 0x0;
    points[0].x._limbs[5] = 0x0;

    points[0].y._limbs[0] = 0x0252450a;
    points[0].y._limbs[1] = 0x3df5ab37;
    points[0].y._limbs[2] = 0x0000035f;
    points[0].y._limbs[3] = 0x0;
    points[0].y._limbs[4] = 0x0;
    points[0].y._limbs[5] = 0x0;

    // POINT B
    points[1].x._limbs[0] = 0x215dc365;
    points[1].x._limbs[1] = 0x834cefb7;
    points[1].x._limbs[2] = 0x00000679;
    points[1].x._limbs[3] = 0x0;
    points[1].x._limbs[4] = 0x0;
    points[1].x._limbs[5] = 0x0;

    points[1].y._limbs[0] = 0x4e6fdfab;
    points[1].y._limbs[1] = 0xbc50388c;
    points[1].y._limbs[2] = 0x00004084;
    points[1].y._limbs[3] = 0x0;
    points[1].y._limbs[4] = 0x0;
    points[1].y._limbs[5] = 0x0;

    parameters->Pmod._limbs[0] = 0xca899cf5;
    parameters->Pmod._limbs[1] = 0x5177412a;
    parameters->Pmod._limbs[2] = 0x000062ce;
    parameters->Pmod._limbs[3] = 0x0;
    parameters->Pmod._limbs[4] = 0x0;
    parameters->Pmod._limbs[5] = 0x0;

    cudaCheckError(hipSetDevice(0));

    cudaCheckError(hipMalloc((void **)&gpuPoints, sizeof(ECC_192_point) * 2));
    cudaCheckError(hipMemcpy(gpuPoints, points, sizeof(ECC_192_point) * 2, hipMemcpyHostToDevice));

    cudaCheckError(hipMalloc((void **)&gpuParameters, sizeof(EC_parameters)));
    cudaCheckError(hipMemcpy(gpuParameters, parameters, sizeof(EC_parameters), hipMemcpyHostToDevice));

    cudaCheckError(cgbn_error_report_alloc(&report));

    kernel_test<<<(INSTANCES + TPI - 1) / TPI, 128>>>(report, gpuPoints, gpuParameters);

    cudaCheckError(hipDeviceSynchronize());

    // CGBN_CHECK(report);

    cudaCheckError(hipMemcpy(points, gpuPoints, sizeof(ECC_192_point) * 2, hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(gpuPoints));
    cudaCheckError(cgbn_error_report_free(report));

    // ASSERT R = P + Q
    REQUIRE(points[0].x._limbs[0] == 0xbdd7ca6a);
    REQUIRE(points[0].x._limbs[1] == 0x142c9e7d);
    REQUIRE(points[0].x._limbs[2] == 0x00005a8b);
    REQUIRE(points[0].x._limbs[3] == 0x0);
    REQUIRE(points[0].x._limbs[4] == 0x0);
    REQUIRE(points[0].x._limbs[5] == 0x0);

    REQUIRE(points[0].y._limbs[0] == 0x7328d137);
    REQUIRE(points[0].y._limbs[1] == 0x489dff15);
    REQUIRE(points[0].y._limbs[2] == 0x000027b6);
    REQUIRE(points[0].y._limbs[3] == 0x0);
    REQUIRE(points[0].y._limbs[4] == 0x0);
    REQUIRE(points[0].y._limbs[5] == 0x0);

    free(points);
}

TEST_CASE("ECC_79p double point [1]")
{
    ECC_192_point *points, *gpuPoints;
    EC_parameters *parameters, *gpuParameters;
    cgbn_error_report_t *report;

    points = (ECC_192_point *)malloc(sizeof(ECC_192_point) * 2);
    parameters = (EC_parameters *)malloc(sizeof(EC_parameters));

    // POINT A
    points[0].x._limbs[0] = 0x8475057d;
    points[0].x._limbs[1] = 0x4b201c20;
    points[0].x._limbs[2] = 0x0000315d;
    points[0].x._limbs[3] = 0x0;
    points[0].x._limbs[4] = 0x0;
    points[0].x._limbs[5] = 0x0;

    points[0].y._limbs[0] = 0x0252450a;
    points[0].y._limbs[1] = 0x3df5ab37;
    points[0].y._limbs[2] = 0x0000035f;
    points[0].y._limbs[3] = 0x0;
    points[0].y._limbs[4] = 0x0;
    points[0].y._limbs[5] = 0x0;

    parameters->Pmod._limbs[0] = 0xca899cf5;
    parameters->Pmod._limbs[1] = 0x5177412a;
    parameters->Pmod._limbs[2] = 0x000062ce;
    parameters->Pmod._limbs[3] = 0x0;
    parameters->Pmod._limbs[4] = 0x0;
    parameters->Pmod._limbs[5] = 0x0;

    parameters->a._limbs[0] = 0xbc45733c;
    parameters->a._limbs[1] = 0x5e6dddb1;
    parameters->a._limbs[2] = 0x000039c9;
    parameters->a._limbs[3] = 0x0;
    parameters->a._limbs[4] = 0x0;
    parameters->a._limbs[5] = 0x0;

    cudaCheckError(hipSetDevice(0));

    cudaCheckError(hipMalloc((void **)&gpuPoints, sizeof(ECC_192_point) * 2));
    cudaCheckError(hipMemcpy(gpuPoints, points, sizeof(ECC_192_point) * 2, hipMemcpyHostToDevice));

    cudaCheckError(hipMalloc((void **)&gpuParameters, sizeof(EC_parameters)));
    cudaCheckError(hipMemcpy(gpuParameters, parameters, sizeof(EC_parameters), hipMemcpyHostToDevice));

    cudaCheckError(cgbn_error_report_alloc(&report));

    kernel_test_double<<<(INSTANCES + TPI - 1) / TPI, 128>>>(report, gpuPoints, gpuParameters);

    cudaCheckError(hipDeviceSynchronize());

    CGBN_CHECK(report);

    cudaCheckError(hipMemcpy(points, gpuPoints, sizeof(ECC_192_point) * 2, hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(gpuPoints));
    cudaCheckError(cgbn_error_report_free(report));

    // ASSERT R = P + Q
    REQUIRE(points[0].x._limbs[0] == 0x47e3e095);
    REQUIRE(points[0].x._limbs[1] == 0x3e221adb);
    REQUIRE(points[0].x._limbs[2] == 0x00004659);
    REQUIRE(points[0].x._limbs[3] == 0x0);
    REQUIRE(points[0].x._limbs[4] == 0x0);
    REQUIRE(points[0].x._limbs[5] == 0x0);

    REQUIRE(points[0].y._limbs[0] == 0x0725a4e3);
    REQUIRE(points[0].y._limbs[1] == 0x42bee392);
    REQUIRE(points[0].y._limbs[2] == 0x000059d6);
    REQUIRE(points[0].y._limbs[3] == 0x0);
    REQUIRE(points[0].y._limbs[4] == 0x0);
    REQUIRE(points[0].y._limbs[5] == 0x0);

    free(points);
}