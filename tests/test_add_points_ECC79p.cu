#include "hip/hip_runtime.h"
//
// Created by atlomak on 05.05.24.
//


#include "catch2/catch_test_macros.hpp"
#include "main.cu"


// Minimal kernel to test device add_points function
__global__ void kernel_test(cgbn_error_report_t *report, ECC_192_point *points, EC_parameters *parameters)
{
    context_t bn_context(cgbn_report_monitor, report, 1); // construct a context
    env192_t bn192_env(bn_context.env<env192_t>());

    env192_t::cgbn_t Pmod;
    dev_ECC_192_point P0, P1;

    cgbn_load(bn192_env, P0.x, &(points[0].x));
    cgbn_load(bn192_env, P0.y, &(points[0].y));

    cgbn_load(bn192_env, P1.x, &(points[1].x));
    cgbn_load(bn192_env, P1.y, &(points[1].y));

    cgbn_load(bn192_env, Pmod, &(parameters->Pmod));

    dev_ECC_192_point result = add_points(bn192_env, P0, P1, Pmod);

    cgbn_store(bn192_env, &(points[0].x), result.x);
    cgbn_store(bn192_env, &(points[0].y), result.y);
}

TEST_CASE("ECC_79p add points [1]")
{
    ECC_192_point *points, *gpuPoints;
    EC_parameters *parameters, *gpuParameters;
    cgbn_error_report_t *report;

    points = (ECC_192_point *)malloc(sizeof(ECC_192_point) * 2);
    parameters = (EC_parameters *)malloc(sizeof(EC_parameters));

    // POINT A
    points[0].x._limbs[0] = 0x8475057d;
    points[0].x._limbs[1] = 0x4b201c20;
    points[0].x._limbs[2] = 0x0000315d;
    points[0].x._limbs[3] = 0x0;
    points[0].x._limbs[4] = 0x0;
    points[0].x._limbs[5] = 0x0;

    points[0].y._limbs[0] = 0x0252450a;
    points[0].y._limbs[1] = 0x3df5ab37;
    points[0].y._limbs[2] = 0x0000035f;
    points[0].y._limbs[3] = 0x0;
    points[0].y._limbs[4] = 0x0;
    points[0].y._limbs[5] = 0x0;

    // POINT B
    points[1].x._limbs[0] = 0x215dc365;
    points[1].x._limbs[1] = 0x834cefb7;
    points[1].x._limbs[2] = 0x00000679;
    points[1].x._limbs[3] = 0x0;
    points[1].x._limbs[4] = 0x0;
    points[1].x._limbs[5] = 0x0;

    points[1].y._limbs[0] = 0x4e6fdfab;
    points[1].y._limbs[1] = 0xbc50388c;
    points[1].y._limbs[2] = 0x00004084;
    points[1].y._limbs[3] = 0x0;
    points[1].y._limbs[4] = 0x0;
    points[1].y._limbs[5] = 0x0;

    parameters->Pmod._limbs[0] = 0xca899cf5;
    parameters->Pmod._limbs[1] = 0x5177412a;
    parameters->Pmod._limbs[2] = 0x000062ce;
    parameters->Pmod._limbs[3] = 0x0;
    parameters->Pmod._limbs[4] = 0x0;
    parameters->Pmod._limbs[5] = 0x0;

    cudaCheckError(hipSetDevice(0));

    cudaCheckError(hipMalloc((void **)&gpuPoints, sizeof(ECC_192_point) * 2));
    cudaCheckError(hipMemcpy(gpuPoints, points, sizeof(ECC_192_point) * 2, hipMemcpyHostToDevice));

    cudaCheckError(hipMalloc((void **)&gpuParameters, sizeof(EC_parameters)));
    cudaCheckError(hipMemcpy(gpuParameters, parameters, sizeof(EC_parameters), hipMemcpyHostToDevice));

    cudaCheckError(cgbn_error_report_alloc(&report));

    kernel_test<<<(INSTANCES + TPI - 1) / TPI, 128>>>(report, gpuPoints, gpuParameters);

    cudaCheckError(hipDeviceSynchronize());

    // CGBN_CHECK(report);

    cudaCheckError(hipMemcpy(points, gpuPoints, sizeof(ECC_192_point) * 2, hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(gpuPoints));
    cudaCheckError(cgbn_error_report_free(report));

    // ASSERT R = P + Q
    REQUIRE(points[0].x._limbs[0] == 0xbdd7ca6a);
    REQUIRE(points[0].x._limbs[1] == 0x142c9e7d);
    REQUIRE(points[0].x._limbs[2] == 0x00005a8b);
    REQUIRE(points[0].x._limbs[3] == 0x0);
    REQUIRE(points[0].x._limbs[4] == 0x0);
    REQUIRE(points[0].x._limbs[5] == 0x0);

    REQUIRE(points[0].y._limbs[0] == 0x7328d137);
    REQUIRE(points[0].y._limbs[1] == 0x489dff15);
    REQUIRE(points[0].y._limbs[2] == 0x000027b6);
    REQUIRE(points[0].y._limbs[3] == 0x0);
    REQUIRE(points[0].y._limbs[4] == 0x0);
    REQUIRE(points[0].y._limbs[5] == 0x0);

    free(points);
}
