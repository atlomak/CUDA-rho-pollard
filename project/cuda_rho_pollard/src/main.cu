#include "hip/hip_runtime.h"
//
// Created by atlomak on 05.05.24.
//

#include "main.cuh"
#include <iostream>

__device__ __host__ int64_t modAdd(int64_t a, int64_t b, int64_t mod)
{
    int64_t result = (a + b) % mod;
    return result < 0 ? result + mod : result;
}

__device__ __host__ int64_t inline modSub(int64_t a, int64_t b, int64_t mod)
{
    int64_t result = (a - b) % mod;
    return result < 0 ? result + mod : result;
}

__device__ __host__ int64_t modMult(int64_t a, int64_t b, int64_t mod)
{
    int64_t result = (a * b) % mod;
    return result < 0 ? result + mod : result;
}

__device__ __host__ int64_t modInv(int64_t a, int64_t mod)
{
    a = a % mod;
    for (int64_t x = 1; x < mod; x++)
    {
        if (modMult(a, x, mod) == 1)
            return x;
    }
    return 1; // Should never happen if mod is prime
}

__device__ __host__ ECC_point mul_point(ECC_point P1, int64_t a, int64_t Pmod)
{
    if (P1.y == 0) // Check for point64_t at infinity
        return (ECC_point){0, 0};

    int64_t s = modMult(modAdd(modMult(3, modMult(P1.x, P1.x, Pmod), Pmod), a, Pmod),
                        modInv(2 * P1.y, Pmod), Pmod);

    int64_t x3 = modSub(modMult(s, s, Pmod), modMult(2, P1.x, Pmod), Pmod);
    int64_t y3 = modSub(modMult(s, modSub(P1.x, x3, Pmod), Pmod), P1.y, Pmod);

    return (ECC_point){x3, y3};
}

__device__ __host__ ECC_point add_points(ECC_point P1, ECC_point P2, int64_t Pmod)
{
    int64_t t1 = modSub(P1.y, P2.y, Pmod); // y1-y2
    int64_t t2 = modSub(P1.x, P2.x, Pmod); // x1-x2
    int64_t t3 = modInv(t2, Pmod); // 1/(x1-x2)

    int64_t s = modMult(t1, t3, Pmod); // t4 = (y1-y) * 1/(x1-x2)

    int64_t t5 = modMult(s, s, Pmod);
    int64_t t6 = modAdd(P1.x, P2.x, Pmod);

    int64_t x3 = modSub(t5, t6, Pmod); // x3 = s^2 - x1 - x2

    int64_t t7 = modMult(-s, x3, Pmod); // -s * x3
    int64_t t8 = modSub(t7, P1.y, Pmod); // (-s * 3) - y1
    int64_t t9 = modMult(s, P1.x, Pmod); // s * x1

    int64_t y3 = modAdd(t8, t9, Pmod); // y3 = (-s * 3) - y1 - s * x1

    return (ECC_point){x3, y3};
}


#ifndef UNIT_TESTING
int64_t main()
{
    std::cout << "Hello thesis";
}
#endif
