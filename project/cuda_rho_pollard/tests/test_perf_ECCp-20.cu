//
// Created by atlomak on 31.05.24.
//

#include <catch2/catch_test_macros.hpp>
#include "main.cuh"
#include "ecc.cuh"

#define FIELD_ORDER 0xD3915
#define CURVE_A_PARAM 0xb44bc
#define CURVE_B_PARAM 0xa999a


TEST_CASE("Kernel add points performance test")
{
    SECTION("Case 1")
    {
        ECC_point p1 = ECC_point{184224, 74658};
        ECC_point p2 = ECC_point{428817, 567437};
        ECC_point result;

        hipEvent_t start, stop;
        cudaCheckError(hipEventCreate(&start))
        cudaCheckError(hipEventCreate(&stop))
        cudaCheckError(hipEventRecord(start, 0))

        ECC_point *dev_a, *dev_b, *dev_result;

        cudaCheckError(hipMalloc((void**)&dev_a, sizeof(ECC_point)))
        cudaCheckError(hipMalloc((void**)&dev_b, sizeof(ECC_point)))
        cudaCheckError(hipMalloc((void**)&dev_result, sizeof(ECC_point)))

        cudaCheckError(hipMemcpy(dev_a, &p1, sizeof(ECC_point), hipMemcpyHostToDevice))
        cudaCheckError(hipMemcpy(dev_b, &p2, sizeof(ECC_point), hipMemcpyHostToDevice))

        test_kernel_add <<< 1, 1 >>>(dev_a, dev_b, dev_result, FIELD_ORDER);

        cudaCheckError(hipMemcpy(&result, dev_result, sizeof(ECC_point), hipMemcpyDeviceToHost))

        cudaCheckError(hipEventRecord(stop, 0));
        cudaCheckError(hipEventSynchronize(stop));
        float elapsedTime;
        cudaCheckError(hipEventElapsedTime(&elapsedTime, start, stop))

        printf("Time to add: %3.1f\n", elapsedTime);

        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_result);

        REQUIRE(result.x == 109605);
        REQUIRE(result.y == 690162);
    }
}